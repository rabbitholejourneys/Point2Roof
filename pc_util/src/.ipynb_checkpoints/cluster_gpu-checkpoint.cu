#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>


#include "cluster_gpu.h"
#include "cuda_utils.h"


__device__ float get_dis(float x1, float y1, float z1, float x2, float y2, float z2) {
	float dis = (x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2) + (z1 - z2) * (z1 - z2);
	return sqrt(dis);
}
/*
__device__ void dfs (int i, int c, int n, int min_pts, const int* pts_cnt, const int* pts_adj, int* idx, int label) {
    idx[i] = c;
    if(pts_cnt[i] < min_pts) return;

    for(int j=0;j<n;j++) {

        int adj = pts_adj[i * n + j];
        printf("%d   %d     %d\n", i * n, i * n + j, adj);
        if (adj == -1) break;
        if (idx[adj] == -1)
            dfs(adj, c, n, min_pts, pts_cnt, pts_adj, idx, label);
    }
}
*/

__global__ void dbscan_kernel_fast(int b, int n, float eps, int min_pts, const float *__restrict__ xyz, int *__restrict__ idx,
    int *__restrict__ pts_cnt, int *__restrict__ pts_adj, int *__restrict__ pts_stack) {
    // xyz: (B, N, 3)
    // output:
    //      idx: (B, N)
    int bs_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (bs_idx >= b) return;

    xyz += bs_idx * n * 3;
    idx += bs_idx * n;
    pts_cnt += bs_idx * n;
    pts_stack += bs_idx * n;
    pts_adj += bs_idx * n * n;

    for(int i=0;i<n;i++) {
        pts_cnt[i] = 0;
        for(int j=0;j<n;j++) {
            pts_adj[i * n + j] = -1;
            if(i==j) continue;
            float x1 = xyz[i * 3 + 0];
            float y1 = xyz[i * 3 + 1];
            float z1 = xyz[i * 3 + 2];
            float x2 = xyz[j * 3 + 0];
            float y2 = xyz[j * 3 + 1];
            float z2 = xyz[j * 3 + 2];

            if(get_dis(x2, y2, z2, -10.0, -10.0, -10.0) < 1e-3) continue;
            if(get_dis(x1, y1, z1, x2, y2, z2) <= eps) {
            pts_adj[i * n + pts_cnt[i]] = j;
                pts_cnt[i] += 1;
            }

        }
    }

    int cluster_idx = 0;

    for(int i=0;i<n;i++) {
        if(idx[i] != -1) continue;

        if(pts_cnt[i] >= min_pts) {
            for(int j=0;j<n;j++)
                pts_stack[j] = -1;
            pts_stack[0] = i;
            int stack_idx = 0;
            int stack_len = 1;
            while (stack_idx < n && pts_stack[stack_idx] != -1)
            {
                int pts_idx = pts_stack[stack_idx];
                idx[pts_idx] = cluster_idx;
                if(pts_cnt[pts_idx] < min_pts){
                    stack_idx += 1;
                    continue;
                }
                for(int j=0;j<n;j++) {
                    int adj = pts_adj[pts_idx * n + j];
                    if (adj == -1) break;
                    if (idx[adj] == -1)
                    {
                        idx[adj] = -2;
                        pts_stack[stack_len++] = adj;
                    }
                }
                stack_idx += 1;
            }
            cluster_idx += 1;
        }
    }
}


void dbscan_kernel_launcher_fast(int b, int n, float eps, int min_pts, const float *xyz, int *idx) {
    // xyz: (B, N, 3)
    // output:
    //      idx: (B, N)

    hipError_t err;

    dim3 blocks(DIVUP(b, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    int* pts_cnt;
    int* pts_stack;
	int* pts_adj;

	err = hipMalloc((void**)&pts_cnt, b * n * sizeof(int));
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }

    err = hipMalloc((void**)&pts_stack, b * n * sizeof(int));
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }

    err = hipMalloc((void**)&pts_adj, b * n * n * sizeof(int));
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }

    dbscan_kernel_fast<<<blocks, threads>>>(b, n, eps, min_pts, xyz, idx, pts_cnt, pts_adj, pts_stack);
    hipDeviceSynchronize();  // for using printf in kernel function
    hipFree(pts_cnt);
    hipFree(pts_stack);
    hipFree(pts_adj);
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}



__global__ void cluster_pts_kernel_fast(int b, int n, int m, const float *__restrict__ xyz, const int *__restrict__ idx,
    float *__restrict__ new_xyz, int *__restrict__ num) {
    int bs_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (bs_idx >= b ) return;

    xyz += bs_idx * n * 3;
    idx += bs_idx * n;
    new_xyz += bs_idx * m * 3;
    num += bs_idx * m;

    for(int i=0;i<n;i++) {
        if (idx[i] == -1) continue;
        int c_idx = idx[i];
        new_xyz[c_idx * 3 + 0] += xyz[i * 3 + 0];
        new_xyz[c_idx * 3 + 1] += xyz[i * 3 + 1];
        new_xyz[c_idx * 3 + 2] += xyz[i * 3 + 2];
        num[c_idx] += 1;
    }
    for(int i=0;i<m;i++) {
        if (num[i] == 0) break;
        new_xyz[i * 3 + 0] /= num[i];
        new_xyz[i * 3 + 1] /= num[i];
        new_xyz[i * 3 + 2] /= num[i];
    }

}




void cluster_pts_kernel_launcher_fast(int b, int n, int m, const float *xyz, const int *idx, float *new_xyz, int *num) {
    hipError_t err;

    dim3 blocks(DIVUP(b, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    cluster_pts_kernel_fast<<<blocks, threads>>>(b, n, m, xyz, idx, new_xyz, num);
    // hipDeviceSynchronize();  // for using printf in kernel function
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}


